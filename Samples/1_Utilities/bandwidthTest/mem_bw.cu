#include "stdio.h"

#include <hip/hip_runtime.h>

#define SIZE (1e9)
#define ITERS (1e3)

int main(void) {

    float* src;
    float* dst;
    hipEvent_t start, stop;
    
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipMalloc((void**)&src, SIZE);
    hipMalloc((void**)&dst, SIZE);

    hipEventRecord(start, 0);

    for (int i = 0; i < ITERS; i++) {
        hipMemcpy((void*)dst, (void*)src, SIZE, hipMemcpyDeviceToDevice);
    }

    hipEventRecord(stop, 0);
    hipDeviceSynchronize();
    
    float ms;
    hipEventElapsedTime(&ms, start, stop);

    float s = ms / 1000.0;
    float mem_transferred = (2.0f * SIZE * ITERS) / (float)1e9;
    float mem_bw = mem_transferred / s;

    printf("s: %f\n", s);
    printf("mem_transferred: %f\n", mem_transferred);
    printf("mem_bw: %f\n", mem_bw);

    return 0;
}
